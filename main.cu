#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <random>
#include <fstream>
#include <cmath>

#ifndef BINARY_OUTPUT
#include <vtkVersion.h>
#include <vtkSmartPointer.h>
#include <vtkXMLStructuredGridWriter.h>
#include <vtkStructuredGrid.h>
#include <vtkPointData.h>
#include <vtkDoubleArray.h>
#endif

// set a 3D volume
// To compile it with nvcc execute: nvcc -O2 -o set3d set3d.cu
//define the data set size (cubic volume)
#define DATAXSIZE 200
#define DATAYSIZE 200
#define DATAZSIZE 200
//define the chunk sizes that each threadblock will work on

using namespace std;

#ifdef BINARY_OUTPUT
void write_output_vtk(double c[][DATAYSIZE][DATAXSIZE], int t, int nx, int ny, int nz, string output, string variableName)
{
    string name = "./out/" + output + variableName+ "_" + to_string(t) + ".bin";
    std::ofstream ofile(name, std::ios::binary);
    ofile.write((char*) c, sizeof(double)*DATAXSIZE*DATAYSIZE*DATAZSIZE);
}
#else
vtkSmartPointer<vtkDoubleArray> convertArrayToVTK(double phi[][DATAYSIZE][DATAXSIZE], char* name)
{

  int counter = 0;
  vtkSmartPointer<vtkDoubleArray> phiVTK =
          vtkSmartPointer<vtkDoubleArray>::New();

  phiVTK->SetNumberOfComponents(1);
  phiVTK->SetNumberOfTuples(DATAXSIZE * DATAYSIZE * DATAZSIZE);

  for (unsigned int idx = 0.0; idx < DATAXSIZE; idx++) {
  for (unsigned int idy = 0.0; idy < DATAYSIZE; idy++) {
  for (unsigned int idz = 0.0; idz < DATAZSIZE; idz++) {

          phiVTK->SetValue(counter, phi[idx][idy][idz]);
          counter++;

  }
  }
  }

  phiVTK->SetName(name);

  return phiVTK;

}

vtkSmartPointer<vtkPoints> createVTKGrid()
{

 vtkSmartPointer<vtkPoints> points =
    vtkSmartPointer<vtkPoints>::New();

  for (unsigned int idx = 0.0; idx < DATAXSIZE; idx++) {
  for (unsigned int idy = 0.0; idy < DATAYSIZE; idy++) {
  for (unsigned int idz = 0.0; idz < DATAZSIZE; idz++) {

   points->InsertNextPoint(idx, idy, idz);

  }
  }
  }

  return points;

}

void writeVTKFile(std::vector<vtkSmartPointer<vtkDoubleArray>> Arrays, vtkSmartPointer<vtkPoints> points, int t)
{

   string name = "./out/output_" + to_string(t) + ".vtk";

   vtkSmartPointer<vtkStructuredGrid> structuredGrid =
    vtkSmartPointer<vtkStructuredGrid>::New();

  structuredGrid->SetDimensions(DATAXSIZE,DATAYSIZE,DATAZSIZE);
  structuredGrid->SetPoints(points);
  for (int i = 0; i < Arrays.size(); i++)
  {
  structuredGrid->GetPointData()->AddArray(Arrays[i]);
  }

  vtkSmartPointer<vtkXMLStructuredGridWriter> writer =
    vtkSmartPointer<vtkXMLStructuredGridWriter>::New();

  writer->SetFileName(name.c_str());

  writer->SetInputData(structuredGrid);

  writer->Update();

}
#endif

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)

void computeIDs(int *IDx, int *IDy, int *IDz)
{
     int counter = 0;
     for (unsigned int idx = 0.0; idx < DATAXSIZE; idx++) {
     for (unsigned int idy = 0.0; idy < DATAYSIZE; idy++) {
      for (unsigned int idz = 0.0; idz < DATAZSIZE; idz++) {
      IDx[counter] = idx;
      IDy[counter] = idy;
      IDz[counter] = idz;
      counter++;
    }
     }
      }
}

__device__ double dFphi(double phi, double u, double lambda)
{

  return (-phi*(1.0-phi*phi)+lambda*u*(1.0-phi*phi)*(1.0-phi*phi));

}

__device__ double GradientX(double phi[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  double phix = (phi[x+1][y][z] - phi[x-1][y][z]) / (2.0*dx);

  return phix;

}

__device__ double GradientY(double phi[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  double phiy = (phi[x][y+1][z] - phi[x][y-1][z]) / (2.0*dy);

  return phiy;

}

__device__ double GradientZ(double phi[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  double phiz = (phi[x][y][z+1] - phi[x][y][z-1]) / (2.0*dz);

  return phiz;

}

__device__ double Divergence(double phix[][DATAYSIZE][DATAXSIZE], double phiy[][DATAYSIZE][DATAXSIZE], double phiz[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  double div = GradientX(phix,dx,dy,dz,x,y,z) + GradientY(phiy,dx,dy,dz,x,y,z) + GradientZ(phiz,dx,dy,dz,x,y,z);

  return div;

}

__device__ double Laplacian(double phi[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  double phixx = (phi[x+1][y][z] + phi[x-1][y][z] - 2.0*phi[x][y][z]) / (dx*dx);
  double phiyy = (phi[x][y+1][z] + phi[x][y-1][z] - 2.0*phi[x][y][z]) / (dy*dy);
  double phizz = (phi[x][y][z+1] + phi[x][y][z-1] - 2.0*phi[x][y][z]) / (dz*dz);

  double result = phixx + phiyy + phizz;

  return result;

}

__device__ double An(double phix, double phiy, double phiz, double epsilon)
{
 if (phix != 0.0 || phiy != 0.0 || phiz != 0.0){
 return ((1.0-3.0*epsilon)*(1.0+(((4.0*epsilon)/(1.0-3.0*epsilon))*((phix*phix*phix*phix+phiy*phiy*phiy*phiy+phiz*phiz*phiz*phiz)/((phix*phix+phiy*phiy+phiz*phiz)*(phix*phix+phiy*phiy+phiz*phiz))))));
 }
 else
 {
 return (1.0-((5.0/3.0)*epsilon));
 }
}

__device__ double Wn(double phix, double phiy, double phiz, double epsilon, double W0)
{

  return (W0*An(phix,phiy,phiz,epsilon));

}

__device__ double taun(double phix, double phiy, double phiz, double epsilon, double tau0)
{

  return (tau0*An(phix,phiy,phiz,epsilon)*An(phix,phiy,phiz,epsilon));

}

__device__ double dFunc(double l, double m, double n)
{
 if (l != 0.0 || m != 0.0 || n != 0.0){
 return (((l*l*l*(m*m+n*n))-(l*(m*m*m*m+n*n*n*n)))/((l*l+m*m+n*n)*(l*l+m*m+n*n)));
 }
 else
 {
 return 0.0;
 }
}

__global__ void calculateForce(double phi[][DATAYSIZE][DATAXSIZE], double Fx[][DATAYSIZE][DATAXSIZE], double Fy[][DATAYSIZE][DATAXSIZE], double Fz[][DATAYSIZE][DATAXSIZE], int *IDx, int *IDy, int *IDz, double dx, double dy, double dz, double epsilon, double W0, double tau0)
{

 unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

 if ((IDx[idx] < (DATAXSIZE-1)) && (IDy[idx] < (DATAYSIZE-1)) && (IDz[idx] < (DATAZSIZE-1)) && (IDx[idx] > (0)) && (IDy[idx] > (0)) && (IDz[idx] > (0))){

  double phix = GradientX(phi,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]);
  double phiy = GradientY(phi,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]);
  double phiz = GradientZ(phi,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]);
  double sqGphi = phix*phix+phiy*phiy+phiz*phiz;

  Fx[IDx[idx]][IDy[idx]][IDz[idx]] = Wn(phix,phiy,phiz,epsilon,W0) *  Wn(phix,phiy,phiz,epsilon,W0) * phix + sqGphi * Wn(phix,phiy,phiz,epsilon,W0) * (16.0*W0*epsilon) * dFunc(phix,phiy,phiz);
  Fy[IDx[idx]][IDy[idx]][IDz[idx]] = Wn(phix,phiy,phiz,epsilon,W0) *  Wn(phix,phiy,phiz,epsilon,W0) * phiy + sqGphi * Wn(phix,phiy,phiz,epsilon,W0) * (16.0*W0*epsilon) * dFunc(phiy,phiz,phix);
  Fz[IDx[idx]][IDy[idx]][IDz[idx]] = Wn(phix,phiy,phiz,epsilon,W0) *  Wn(phix,phiy,phiz,epsilon,W0) * phiz + sqGphi * Wn(phix,phiy,phiz,epsilon,W0) * (16.0*W0*epsilon) * dFunc(phiz,phix,phiy);
 }
 else
 {
  Fx[IDx[idx]][IDy[idx]][IDz[idx]] = 0.0;
  Fy[IDx[idx]][IDy[idx]][IDz[idx]] = 0.0;
  Fz[IDx[idx]][IDy[idx]][IDz[idx]] = 0.0;
 }

}

// device function to set the 3D volume
__global__ void allenCahn(double phinew[][DATAYSIZE][DATAXSIZE], double phiold[][DATAYSIZE][DATAXSIZE], double uold[][DATAYSIZE][DATAXSIZE], double Fx[][DATAYSIZE][DATAXSIZE], double Fy[][DATAYSIZE][DATAXSIZE], double Fz[][DATAYSIZE][DATAXSIZE], int *IDx, int *IDy, int *IDz, double epsilon, double W0, double tau0, double lambda, double dt, double dx, double dy, double dz)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

    if ((IDx[idx] < (DATAXSIZE-1)) && (IDy[idx] < (DATAYSIZE-1)) && (IDz[idx] < (DATAZSIZE-1)) && (IDx[idx] > (0)) && (IDy[idx] > (0)) && (IDz[idx] > (0))){

      double phix = GradientX(phiold,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]);
      double phiy = GradientY(phiold,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]);
      double phiz = GradientZ(phiold,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]); 
  
      phinew[IDx[idx]][IDy[idx]][IDz[idx]] = phiold[IDx[idx]][IDy[idx]][IDz[idx]] + (dt / taun(phix,phiy,phiz,epsilon,tau0)) * (Divergence(Fx,Fy,Fz,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]) - dFphi(phiold[IDx[idx]][IDy[idx]][IDz[idx]],uold[IDx[idx]][IDy[idx]][IDz[idx]],lambda));
      }
}

__global__ void boundaryConditionsPhi(double phinew[][DATAYSIZE][DATAXSIZE], int *IDx, int *IDy, int *IDz)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (IDx[idx] == 0){
      phinew[IDx[idx]][IDy[idx]][IDz[idx]] = -1.0;
      }
    else if (IDx[idx] == DATAXSIZE-1){
      phinew[IDx[idx]][IDy[idx]][IDz[idx]] = -1.0;
      }
    else if (IDy[idx] == 0){
      phinew[IDx[idx]][IDy[idx]][IDz[idx]] = -1.0;
      }
    else if (IDy[idx] == DATAYSIZE-1){
      phinew[IDx[idx]][IDy[idx]][IDz[idx]] = -1.0;
      }
    else if (IDz[idx] == 0){
      phinew[IDx[idx]][IDy[idx]][IDz[idx]] = -1.0;
      }
    else if (IDz[idx] == DATAZSIZE-1){
      phinew[IDx[idx]][IDy[idx]][IDz[idx]] = -1.0;
      }

}

__global__ void thermalEquation(double unew[][DATAYSIZE][DATAXSIZE], double uold[][DATAYSIZE][DATAXSIZE], double phinew[][DATAYSIZE][DATAXSIZE], double phiold[][DATAYSIZE][DATAXSIZE], int *IDx, int *IDy, int *IDz, double D, double dt, double dx, double dy, double dz)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

    if ((IDx[idx] < (DATAXSIZE-1)) && (IDy[idx] < (DATAYSIZE-1)) && (IDz[idx] < (DATAZSIZE-1)) && (IDx[idx] > (0)) && (IDy[idx] > (0)) && (IDz[idx] > (0))){
      unew[IDx[idx]][IDy[idx]][IDz[idx]] = uold[IDx[idx]][IDy[idx]][IDz[idx]] + 0.5*(phinew[IDx[idx]][IDy[idx]][IDz[idx]]-phiold[IDx[idx]][IDy[idx]][IDz[idx]]) + dt * D * Laplacian(uold,dx,dy,dz,IDx[idx],IDy[idx],IDz[idx]);
      }
}

__global__ void boundaryConditionsU(double unew[][DATAYSIZE][DATAXSIZE], double delta, int *IDx, int *IDy, int *IDz)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (IDx[idx] == 0){
      unew[IDx[idx]][IDy[idx]][IDz[idx]] =  -delta;
      }
    else if (IDx[idx] == DATAXSIZE-1){
      unew[IDx[idx]][IDy[idx]][IDz[idx]] =  -delta;
      }
    else if (IDy[idx] == 0){
      unew[IDx[idx]][IDy[idx]][IDz[idx]] =  -delta;
      }
    else if (IDy[idx] == DATAYSIZE-1){
      unew[IDx[idx]][IDy[idx]][IDz[idx]] =  -delta;
      }
    else if (IDz[idx] == 0){
      unew[IDx[idx]][IDy[idx]][IDz[idx]] =  -delta;
      }
    else if (IDz[idx] == DATAZSIZE-1){
      unew[IDx[idx]][IDy[idx]][IDz[idx]] =  -delta;
      }

}

__global__ void Swap(double cnew[][DATAYSIZE][DATAXSIZE], double cold[][DATAYSIZE][DATAXSIZE], int *IDx, int *IDy, int *IDz)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

    double tmp;    

    if ((IDx[idx] < (DATAXSIZE)) && (IDy[idx] < (DATAYSIZE)) && (IDz[idx] < (DATAZSIZE))) {
     tmp=cnew[IDx[idx]][IDy[idx]][IDz[idx]];
     cnew[IDx[idx]][IDy[idx]][IDz[idx]]=cold[IDx[idx]][IDy[idx]][IDz[idx]];
     cold[IDx[idx]][IDy[idx]][IDz[idx]]=tmp;
    }

}

void initializationPhi(double phi[][DATAYSIZE][DATAXSIZE], double r0)
{
    for (unsigned int idx = 0.0; idx < DATAXSIZE; idx++) {
     for (unsigned int idy = 0.0; idy < DATAYSIZE; idy++) {
      for (unsigned int idz = 0.0; idz < DATAZSIZE; idz++) {
      double r = std::sqrt((idx-0.5*DATAXSIZE)*(idx-0.5*DATAXSIZE) + (idy-0.5*DATAYSIZE)*(idy-0.5*DATAYSIZE) + (idz-0.5*DATAZSIZE)*(idz-0.5*DATAZSIZE));
      if (r < r0){
      phi[idx][idy][idz] = 1.0;
      }
      else
      {
      phi[idx][idy][idz] = -1.0;
      }
    }
     }
      }
}

void initializationU(double u[][DATAYSIZE][DATAXSIZE], double r0, double delta)
{
    for (unsigned int idx = 0.0; idx < DATAXSIZE; idx++) {
     for (unsigned int idy = 0.0; idy < DATAYSIZE; idy++) {
      for (unsigned int idz = 0.0; idz < DATAZSIZE; idz++) {
      double r = std::sqrt((idx-0.5*DATAXSIZE)*(idx-0.5*DATAXSIZE) + (idy-0.5*DATAYSIZE)*(idy-0.5*DATAYSIZE) + (idz-0.5*DATAZSIZE)*(idz-0.5*DATAZSIZE));
      if (r < r0) {
      u[idx][idy][idz] = 0.0;
      }
      else
      {
      u[idx][idy][idz] = -delta * (1.0 - std::exp(-(r-r0)));
      }
    }
     }
      }
}

int main(int argc, char *argv[])
{
    double dx = 0.4;
    double dy = 0.4;
    double dz = 0.4;
    double dt = 0.01;
    int t_f = 6000;
    int t_freq = 100;
    double delta = 0.8;
    double r0 = 5.0;
    double epsilon = 0.07;
    double W0 = 1.0;
    double beta0 = 0.0;
    double D = 2.0;
    double d0 = 0.5;
    double a1 = 1.25 / std::sqrt(2.0);
    double a2 = 0.64;
    double lambda = (W0*a1)/(d0);
    double tau0 = ((W0*W0*W0*a1*a2)/(d0*D)) + ((W0*W0*beta0)/(d0));
#ifndef BINARY_OUTPUT
    vtkSmartPointer<vtkPoints> points = createVTKGrid();
#endif
    hipSetDevice(0.0);
    typedef double nRarray[DATAYSIZE][DATAXSIZE];
    const int BLOCK_SIZE = 1024;
    const int siteCount = DATAXSIZE*DATAYSIZE*DATAZSIZE;
    const int GRID_SIZE = (siteCount + BLOCK_SIZE - 1) / BLOCK_SIZE;
// overall data set sizes
    const int nx = DATAXSIZE;
    const int ny = DATAYSIZE;
    const int nz = DATAZSIZE;
// pointers for data set storage via malloc
    nRarray *phi_host; // storage for result stored on host
    nRarray *u_host;
    int *IDx_host;
    int *IDy_host;
    int *IDz_host;
    nRarray *d_phiold;  // storage for result computed on device
    nRarray *d_phinew;
    nRarray *d_uold;
    nRarray *d_unew;
    nRarray *d_Fx;
    nRarray *d_Fy;
    nRarray *d_Fz;
    int *d_IDx;
    int *d_IDy;
    int *d_IDz;
// allocate storage for data set
    hipHostAlloc((void**)&phi_host,(nx*ny*nz)*sizeof(double),hipHostMallocMapped);
    cudaCheckErrors("Failed to allocate host buffer");
    hipHostAlloc((void**)&u_host,(nx*ny*nz)*sizeof(double),hipHostMallocMapped);
    cudaCheckErrors("Failed to allocate host buffer");
    hipHostAlloc((void**)&IDx_host,(nx*ny*nz)*sizeof(double),hipHostMallocMapped);
    cudaCheckErrors("Failed to allocate host buffer");
    hipHostAlloc((void**)&IDy_host,(nx*ny*nz)*sizeof(double),hipHostMallocMapped);
    cudaCheckErrors("Failed to allocate host buffer");
    hipHostAlloc((void**)&IDz_host,(nx*ny*nz)*sizeof(double),hipHostMallocMapped);
    cudaCheckErrors("Failed to allocate host buffer");
    //if ((phi_host = (nRarray *)malloc((nx*ny*nz)*sizeof(double))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
    //if ((u_host = (nRarray *)malloc((nx*ny*nz)*sizeof(double))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
    //if ((IDx_host = (int *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
    //if ((IDy_host = (int *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
    //if ((IDz_host = (int *)malloc((nx*ny*nz)*sizeof(int))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
// allocate GPU device buffers
    hipMalloc((void **) &d_phiold, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_phinew, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_uold, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_unew, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_Fx, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_Fy, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_Fz, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_IDx, (nx*ny*nz)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_IDy, (nx*ny*nz)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_IDz, (nx*ny*nz)*sizeof(int));
    cudaCheckErrors("Failed to allocate device buffer");
// compute result

    initializationPhi(phi_host,r0);
    initializationU(u_host,r0,delta);

#ifdef BINARY_OUTPUT
    write_output_vtk(phi_host,0,nx,ny,nz,"output","phi");
    write_output_vtk(u_host,0,nx,ny,nz,"output","u");
#else
    std::vector<vtkSmartPointer<vtkDoubleArray>> ArraysInitial;

    ArraysInitial.push_back(convertArrayToVTK(phi_host,"phi"));
    ArraysInitial.push_back(convertArrayToVTK(u_host,"u"));

    writeVTKFile(ArraysInitial,points,0);
#endif

    hipMemcpyAsync(d_phiold, phi_host, ((nx*ny*nz)*sizeof(double)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    hipMemcpyAsync(d_uold, u_host, ((nx*ny*nz)*sizeof(double)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    computeIDs(IDx_host,IDy_host,IDz_host);

    hipMemcpyAsync(d_IDx, IDx_host, ((nx*ny*nz)*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    hipMemcpyAsync(d_IDy, IDy_host, ((nx*ny*nz)*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    hipMemcpyAsync(d_IDz, IDz_host, ((nx*ny*nz)*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    double clock_d = double(clock()) / CLOCKS_PER_SEC;

    int t = 0;

    while (t <= t_f) {

    printf("Timestep is: %d\n",t);

    calculateForce<<<GRID_SIZE,BLOCK_SIZE>>>(d_phiold,d_Fx,d_Fy,d_Fz,d_IDx,d_IDy,d_IDz,dx,dy,dz,epsilon,W0,tau0);
    cudaCheckErrors("Kernel launch failure");
    allenCahn<<<GRID_SIZE,BLOCK_SIZE>>>(d_phinew,d_phiold,d_uold,d_Fx,d_Fy,d_Fz,d_IDx,d_IDy,d_IDz,epsilon,W0,tau0,lambda,dt,dx,dy,dz);
    cudaCheckErrors("Kernel launch failure");
    boundaryConditionsPhi<<<GRID_SIZE,BLOCK_SIZE>>>(d_phinew,d_IDx,d_IDy,d_IDz);
    cudaCheckErrors("Kernel launch failure");

    thermalEquation<<<GRID_SIZE,BLOCK_SIZE>>>(d_unew,d_uold,d_phinew,d_phiold,d_IDx,d_IDy,d_IDz,D,dt,dx,dy,dz);
    cudaCheckErrors("Kernel launch failure");
    boundaryConditionsU<<<GRID_SIZE,BLOCK_SIZE>>>(d_unew,delta,d_IDx,d_IDy,d_IDz);
    cudaCheckErrors("Kernel launch failure");

    if (t % t_freq == 0 && t > 0) {

     hipMemcpyAsync(phi_host, d_phinew, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);
     cudaCheckErrors("CUDA memcpy failure");

     hipMemcpyAsync(u_host, d_unew, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);
     cudaCheckErrors("CUDA memcpy failure");

#ifdef BINARY_OUTPUT
    write_output_vtk(phi_host,t,nx,ny,nz,"output","phi");
    write_output_vtk(u_host,t,nx,ny,nz,"output","u");
#else
     std::vector<vtkSmartPointer<vtkDoubleArray>> Arrays;

     Arrays.push_back(convertArrayToVTK(phi_host,"phi"));
     Arrays.push_back(convertArrayToVTK(u_host,"u"));

     writeVTKFile(Arrays,points,t);
#endif

    }
    
    Swap<<<GRID_SIZE,BLOCK_SIZE>>>(d_phinew, d_phiold,d_IDx,d_IDy,d_IDz);
    cudaCheckErrors("Kernel launch failure");

    Swap<<<GRID_SIZE,BLOCK_SIZE>>>(d_unew, d_uold,d_IDx,d_IDy,d_IDz);
    cudaCheckErrors("Kernel launch failure");

    t++;

    }

    hipDeviceSynchronize();
    clock_d = double(clock()) / CLOCKS_PER_SEC - clock_d; 
    printf("GPU time = %.3fms\n",clock_d*1e3);

    free(phi_host);
    free(u_host);
    free(IDx_host);
    free(IDy_host);
    free(IDz_host);
    hipFree(d_phiold);
    cudaCheckErrors("hipFree fail");
    hipFree(d_phinew);
    cudaCheckErrors("hipFree fail");
    hipFree(d_uold);
    cudaCheckErrors("hipFree fail");
    hipFree(d_unew);
    cudaCheckErrors("hipFree fail");
    hipFree(d_Fx);
    cudaCheckErrors("hipFree fail");
    hipFree(d_Fy);
    cudaCheckErrors("hipFree fail");
    hipFree(d_Fz);
    cudaCheckErrors("hipFree fail");
    hipFree(d_IDx);
    cudaCheckErrors("hipFree fail");
    hipFree(d_IDy);
    cudaCheckErrors("hipFree fail");
    hipFree(d_IDz);
    cudaCheckErrors("hipFree fail");
    return 0;
}
